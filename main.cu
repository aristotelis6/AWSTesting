#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int a,int b,int*c)
{
    *c=a+b;
}

int main()
{
    int*dc,*hc;
    hc=new int;
    hipMalloc(&dc,sizeof(int));
    add<<<1,1>>>(2,3,dc);
    hipMemcpy(hc,dc,sizeof(int),hipMemcpyDeviceToHost);
    std::cout<<*hc<<std::endl;
    hipFree(dc);
}